#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include </home/ksugumar/project/headers/helper_functions.h>
#include </home/ksugumar/project/headers/hip/hip_runtime_api.h>
#include ""
#include <chrono>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "/home/ksugumar/project/headers/stb_image.h"
#include "/home/ksugumar/project/headers/stb_image_write.h"

using namespace std;
using namespace chrono;

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Kernel function for image filtering using convolution
__global__ void imageFilteringkernel\
	(const float *d_image_pad, const unsigned int r_pad, const unsigned int c_pad, const float *d_filter,
	const int padding_size, float *d_conv_image, const unsigned int rows, const unsigned int cols)

{
	unsigned int filter_size = 2 * padding_size + 1;

	//Determine the pixel co-ordinates
	const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x + padding_size;
	const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y + padding_size;

	//Multiply and add operation for pixel (j,i)
	if (j > padding_size && j < c_pad - padding_size && i > padding_size && i < r_pad - padding_size)
	{
		unsigned int conv_pixel_pos = (i - padding_size) * cols + (j - padding_size);
		d_conv_image[conv_pixel_pos] = 0.0;
		for (int k = -padding_size; k <= padding_size; k++)
		{
			for (int l = -padding_size; l <= padding_size; l++)
			{
				unsigned int img_pixel_pos = (i + k) * c_pad + (j + l);
				unsigned int filter_pos = (k + padding_size) * filter_size + (l + padding_size);
				d_conv_image[conv_pixel_pos] += d_image_pad[img_pixel_pos] * d_filter[filter_pos];
			}
		}
	}
}

inline unsigned int iDivUp(const unsigned int &a, const unsigned int &b) { return (a%b != 0) ? (a / b + 1) : (a / b); }

int main(int argc, char** argv)
{
	// Read the image file on host
	int rows, cols, bpp;
	uint8_t* h_original_image = stbi_load(argv[1], &cols, &rows, &bpp, 1);
	cout << "The number of rows is " << rows << "\n";
	cout << "The number of columns is " << cols << "\n";
	
	// Declare Image variables
	int padding_size = 2;
	unsigned int r_pad = rows + 2 * padding_size;
	unsigned int c_pad = cols + 2 * padding_size;
	int imsize = rows*cols;
	int imsize_pad = r_pad*c_pad;

	//Allocate space on host for padded input image
	float **h_padded_image;
	h_padded_image = new float*[r_pad];
	for (int i = 0; i < r_pad; i++)
	{
		h_padded_image[i] = new float[c_pad];
	}

	// Fill the 2D array with zeros
	for (int i = 0; i < r_pad; i++)
	{
		for (int j = 0; j < c_pad; j++)
		{
			h_padded_image[i][j] = 0;
		}
	}

	// Copy pixels from the original image to the 2D array, without affecting the padded 0
	for (int i = padding_size; i < r_pad - padding_size; i++)
	{
		for (int j = padding_size; j < c_pad - padding_size; j++)
		{
			h_padded_image[i][j] = *(h_original_image + ((i - padding_size)*cols) + (j - padding_size));
		}
	}

	// Convert the padded image to a 1D array. Accessing 1D arrays are more efficient in GPUs
	float *h_padded_image_1d = new float[imsize_pad];
	for (int q = 0; q < r_pad; q++)
	{
		for (int t = 0; t < c_pad; t++)
		{
			h_padded_image_1d[q * r_pad + t] = h_padded_image[q][t];
		}
	}

	// delete the original 2D padded image after reshaping it to 1D
	delete h_padded_image;

	// Initialize the kernel to be used for convolution as a 1D array

	// Gaussian blur filter 5x5
	float h_filter[25] = { 1, 4,  6,  4, 1, \
		4, 16, 24, 16, 4, \
		6, 24, 36, 24, 6, \
		4, 16, 24, 16, 4, \
		1, 4,  6,  4, 1, };

	for (int f = 0; f < 25; f++) {
		h_filter[f] /= 256.0;
	}
	
//	float h_filter[9] = { -1, -1, -1, \
//			     - 1,  8, -1, \
//			     - 1, -1, -1 };
//
	//float h_filter[9] = {0, 0, 0, \
	//	0,  1, 0, \
	//	0, 0, 0 };

	// Initialize a 1D array to hold the convoluted image
	float *h_conv_image_1d = new float[imsize];

	unsigned int filter_size = 2 * padding_size + 1;

// MEMORY ALLOCATION ON DEVICE STARTS HERE

	//Allocate memory on device for image and transfer image from host to device
	float *d_padded_image;
	unsigned int d_imsize_pad = r_pad * c_pad * sizeof(float);
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_padded_image), d_imsize_pad));
	

	auto h_start = steady_clock::now();
	checkCudaErrors(hipMemcpy(d_padded_image, h_padded_image_1d, d_imsize_pad, hipMemcpyHostToDevice));

	//Allocate memory on device for filter and transfer filter from host to device
	float *d_filter;
	unsigned int d_filtersize = filter_size * filter_size * sizeof(float);
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_filter), d_filtersize));
	checkCudaErrors(hipMemcpy(d_filter, h_filter, d_filtersize, hipMemcpyHostToDevice));

	//Set up the grid and block dimensions for execution
	const unsigned int block_col = 32;
	const unsigned int block_row = 32;
	const dim3 grid( iDivUp(cols, block_col), iDivUp(rows, block_row));
	const dim3 threadBlock(block_col, block_row);

	//Memory allocation for filtered image
	float *d_conv_image;
	unsigned int conv_imsize = rows * cols * sizeof(float);
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_conv_image), conv_imsize));


// **** CONVOLUTION STARTS HERE ! ****
	
	float elapsed = 0;
	hipEvent_t start, stop;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	HANDLE_ERROR(hipEventRecord(start, 0));

	checkCudaErrors(hipDeviceSynchronize());
	imageFilteringkernel <<<grid, threadBlock>>>(d_padded_image, r_pad, c_pad, d_filter, padding_size, d_conv_image, rows, cols);
	checkCudaErrors(hipDeviceSynchronize());

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	cout << "Total Elapsed Time for the Kernel(GPU): " << elapsed << " ms" << endl;

	checkCudaErrors(hipMemcpy(h_conv_image_1d, d_conv_image, conv_imsize, hipMemcpyDeviceToHost));

	auto h_end = steady_clock::now();
	
	cout << "Total Elapsed Time(including data transfer): " << (duration<double>\
					(h_end - h_start).count())*1000.0 << " ms\n" << endl;

	
// **** CONVOLUTION ENDS HERE ! ****
	
	static uint8_t conv_image_final[1024][1024];
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			conv_image_final[i][j] = 0;
		}
	}

	// perform convertion of 1d to 2d
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			int pixel = h_conv_image_1d[i*rows + j];
			if (pixel > 255)
				conv_image_final[i][j] = 255;
			else if (pixel < 0)
				conv_image_final[i][j] = 0;
			else
				conv_image_final[i][j] = pixel;

		}
	}

	// Write convoluted image to file
	stbi_write_jpg(argv[2], cols, rows, 1, conv_image_final, cols);

	return 0;
}
